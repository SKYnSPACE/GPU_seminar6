
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define N (65535)
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__))

static void HandleError( hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString (err ), file, line);
		exit( EXIT_FAILURE );
	}
}

double get_time()
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (double)tv.tv_sec + (double)1e-6 * tv.tv_usec;
}

__global__ void add (int *a, int *b, int *c)
{
	/* PLEASE COMPLETE THE CODE */
}

void add_cpu (int *a, int *b, int *c)
{
	/* PLEASE COMPLETE THE CODE */
}

int main(void)
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	double start_time, end_time;

	//INITIALIZE THE ARRAY
	for (int i=0; i<N; i++)
	{
		a[i] = i;
		b[i] = i*i;
	}

	//GPU MEMORY ALLOCATION for dev_a, dev_b, and dev_c.
	/* PLEASE COMPLETE THE CODE */

	//COPY a and b to ALLOCATED GPU MEMORY
	/* PLEASE COMPLETE THE CODE */

	//GPU CALCULATION TAKES PLACE
	start_time = get_time();
	/* PLEASE COMPLETE THE CODE */
	end_time = get_time();
	printf("[GPU]: %f sec Elpased\n", end_time - start_time);
	
	//COPY c to HOST
	/* PLEASE COMPLETE THE CODE */
	
	//CHECK FOR THE INTEGRITY
	bool success = true;
	for (int i=0; i<N; i++)
	{
		if ((a[i] + b[i])!=c[i])
		{
			printf("%d + %d != %d\n", a[i], b[i], c[i]);
			success = false;
		}
	}

	if (success)
	{
		printf("GPU TEST SUCCESSFUL!\n");
	}

	//FREE the CUDA MEMORY
	/* PLEASE COMPLETE THE CODE */
	
	//CPU CALCULATION TAKES PLACE
	start_time = get_time();
	add_cpu(a, b, c);
	end_time = get_time();
	printf("[CPU]: %f sec Elpased\n", end_time - start_time);

	//CHECK FOR THE INTEGRITY
	for (int i=0; i<N; i++)
	{
		if ((a[i] + b[i])!=c[i])
		{
			printf("%d + %d != %d\n", a[i], b[i], c[i]);
			success = false;
		}
	}

	if (success)
	{
		printf("CPU TEST SUCCESSFUL!\n");
	}
	
}
